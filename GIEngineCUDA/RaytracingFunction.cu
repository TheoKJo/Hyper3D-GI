#include "hip/hip_runtime.h"

// warning at 949 code page
#pragma warning( disable : 4819 ) 

#include <hip/hip_runtime.h>


#include "cuKDTree.h"
#include "cuKDTreeNode.h"
#include "../GIEngineCore/Global.h"

// 32 * 32 = 1024 <= 1024 (GTS 450)
const int BLOCK_SIZE_X = 32;
const int BLOCK_SIZE_Y = 32;

__global__ void ShootRay( GIHit *outHits, RtTriAccel *TriAccelArray, cuKDTree *KDTree, const GIRay *RayArray )
{
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int WIDTH = BLOCK_SIZE_X * gridDim.x;

	int index = tx + bx * BLOCK_SIZE_X
		+ (ty + by * BLOCK_SIZE_Y) * WIDTH;
	
	const GIRay &Ray = RayArray[index];
	GIHit &result = outHits[index];

	result.triNum = 1;
	result.dist = 3.141592f;
}

extern "C" __host__ void cuShootRays( cuKDTree *DeviceKDTree, unsigned int RayCount, const GIRay *RayArray, GIHit *outHits )
{
	GIRay *DeviceRayArray;
	hipMalloc( &DeviceRayArray, RayCount * sizeof(GIRay) );
	hipMemcpy( DeviceRayArray, RayArray, RayCount * sizeof(GIRay), hipMemcpyHostToDevice );

	GIHit *DeviceHitArray;
	hipMalloc( &DeviceHitArray, RayCount * sizeof(GIHit) );

	//hipMemcpy( RayArray, DeviceRayArray, RayCount * sizeof(cuKDTreeNode), hipMemcpyHostToDevice );
	

	// TODO: ũ�� �� ������!! ����� �����ϰ�!!
	dim3 dimGrid( RayCount/BLOCK_SIZE_X, 1, 1 );
	dim3 dimBlock( BLOCK_SIZE_X, 1, 1 );

	ShootRay<<<dimGrid, dimBlock>>>( DeviceHitArray, NULL, DeviceKDTree, DeviceRayArray );
	
	hipMemcpy( outHits, DeviceHitArray, RayCount * sizeof(GIHit), hipMemcpyDeviceToHost );

	hipFree( DeviceRayArray );
	hipFree( DeviceHitArray );
}

