#include "hip/hip_runtime.h"

// warning at 949 code page
#pragma warning( disable : 4819 ) 

#include <hip/hip_runtime.h>
#include "../GIEngineCore/Global.h"

// 32 * 32 = 1024 <= 1024 (GTS 450)
const int BLOCK_SIZE_X = 32;
const int BLOCK_SIZE_Y = 32;

class KDTreeGPU;

__global__ void ShootRayCUDA( GIHit *outHits, RtTriAccel *TriAccelArray, KDTreeGPU *KDTree, const GIRay *RayArray )
{
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int WIDTH = BLOCK_SIZE_X * gridDim.x;

	int index = tx + bx * BLOCK_SIZE_X
		+ (ty + by * BLOCK_SIZE_Y) * WIDTH;
	
	const GIRay &Ray = RayArray[index];
	GIHit &result = outHits[index];
}

__host__ void ShootRays( GIHit *outHits, RtTriAccel *TriAccelArray, KDTreeGPU *KDTree, const GIRay *RayArray, unsigned int RayCount )
{
	// TODO: ũ�� �� ������!! ����� �����ϰ�!!
	dim3 dimGrid( RayCount/BLOCK_SIZE_X, 1, 1 );
	dim3 dimBlock( BLOCK_SIZE_X, 1, 1 );

	ShootRayCUDA<<<dimGrid, dimBlock>>>( outHits, TriAccelArray, KDTree, RayArray );
}