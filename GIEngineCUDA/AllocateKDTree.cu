
#include "cuKDTree.h"
#include "cuKDTreeNode.h"

extern "C" __host__ cuKDTree* CreateDeviceKDTreeAllocateKDTreeMemory( unsigned int NodeCount, cuKDTreeNode *NodeArray, 
															  unsigned int TriangleCount, cuKDTreeTriangle *TriangleArray, 
															  unsigned int TriangleIndexCount, unsigned int *TriangleIndexArray )
{
	cuKDTree *DeviceKDTree = new cuKDTree();

	DeviceKDTree->NodeCount = NodeCount;
	DeviceKDTree->TriangleCount = TriangleCount;
	DeviceKDTree->TriangleIndexCount = TriangleIndexCount;

	// TODO: Error Check

	hipError_t result;

	// node array
	DeviceKDTree->NodeArray = NULL;
	result = hipMalloc( &DeviceKDTree->NodeArray, NodeCount * sizeof(cuKDTreeNode) );
	if( result != hipSuccess )
	{
		int a = result;
		return NULL;
	}
	result = hipMemcpy( DeviceKDTree->NodeArray, NodeArray, NodeCount * sizeof(cuKDTreeNode), hipMemcpyHostToDevice );
	if( result != hipSuccess )
	{
		int a = result;
		return NULL;
	}

	// triangles
	DeviceKDTree->TriangleArray = NULL;
	result = hipMalloc( &DeviceKDTree->TriangleArray, TriangleCount * sizeof(cuKDTreeTriangle) );
	if( result != hipSuccess )
	{
		int a = result;
		return NULL;
	}
	result = hipMemcpy( DeviceKDTree->TriangleArray, TriangleArray, TriangleCount * sizeof(cuKDTreeTriangle), hipMemcpyHostToDevice );
	if( result != hipSuccess )
	{
		int a = result;
		return NULL;
	}

	// triangle indices
	DeviceKDTree->TriangleIndexArray = NULL;
	result = hipMalloc( &DeviceKDTree->TriangleIndexArray, TriangleIndexCount * sizeof(unsigned int) );
	if( result != hipSuccess )
	{
		int a = result;
		return NULL;
	}
	result = hipMemcpy( DeviceKDTree->TriangleIndexArray, TriangleIndexArray, TriangleIndexCount * sizeof(unsigned int), hipMemcpyHostToDevice );
	if( result != hipSuccess )
	{
		int a = result;
		return NULL;
	}


	return DeviceKDTree;
}